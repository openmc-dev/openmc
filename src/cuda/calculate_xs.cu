#include "hip/hip_runtime.h"
#include "openmc/cuda/calculate_xs.h"
#include "openmc/geometry.h" // find_cell
#include "openmc/search.h"

namespace openmc {
namespace gpu {

__constant__ unique_ptr<Material>* materials;
__constant__ unique_ptr<Nuclide>* nuclides;
__constant__ Particle* particles;
__constant__ NuclideMicroXS* micros;
__constant__ double energy_min_neutron;
__constant__ double energy_max_neutron;
__constant__ double log_spacing;
__constant__ unsigned number_nuclides;
__constant__ bool need_depletion_rx;

__managed__ unsigned managed_calculate_fuel_queue_index;
__managed__ unsigned managed_calculate_nonfuel_queue_index;

__global__ void process_calculate_xs_events_device(
  EventQueueItem* __restrict__ queue, unsigned queue_size)
{
  unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= queue_size)
    return;
  Particle* __restrict__ p = particles + queue[tid].idx;
  auto const E = __ldg(&queue[tid].E);
  auto const mat_idx = __ldg(&queue[tid].material);

  // Store pre-collision particle properties
  p->wgt_last_ = p->wgt_;
  p->E_last_ = E;
  p->u_last_ = p->u();
  p->r_last_ = p->r();

  // Reset event variables
  p->event_ = TallyEvent::KILL;
  p->event_nuclide_ = NUCLIDE_NONE;
  p->event_mt_ = REACTION_NONE;

  p->macro_xs_.total = 0.0;
  p->macro_xs_.absorption = 0.0;
  p->macro_xs_.fission = 0.0;
  p->macro_xs_.nu_fission = 0.0;

  // Skip void material
  if (mat_idx == -1)
    return;

  Material const& m = *materials[mat_idx];

  unsigned i_log_union = std::log(E / energy_min_neutron) / log_spacing;

  // Add contribution from each nuclide in material
  auto const n_nuclides = m.nuclide_.size();
  for (int i = 0; i < n_nuclides; ++i) {
    auto const& i_nuclide = m.nuclide_[i];
    auto* __restrict__ micro {
      &micros[number_nuclides * queue[tid].idx + i_nuclide]};

    if (E != micro->last_E || p->sqrtkT_ != micro->last_sqrtkT) {
      auto const& nuclide = *nuclides[i_nuclide];
      micro->elastic = CACHE_INVALID;
      micro->thermal = 0.0;
      micro->thermal_elastic = 0.0;

      // Find the appropriate temperature index. why would someone use
      // nearest?
      double kT = p->sqrtkT_ * p->sqrtkT_;
      double f;

      int i_temp = -1;

      // Find temperatures that bound the actual temperature
      for (i_temp = 0; i_temp < nuclide.kTs_.size() - 1; ++i_temp) {
        if (nuclide.kTs_[i_temp] <= kT && kT < nuclide.kTs_[i_temp + 1])
          break;
      }

      // Randomly sample between temperature i and i+1
      f = (kT - nuclide.kTs_[i_temp]) /
          (nuclide.kTs_[i_temp + 1] - nuclide.kTs_[i_temp]);
      if (f > prn(p->seeds_))
        ++i_temp;

      const auto& grid {nuclide.grid_[i_temp]};
      int i_grid;
      if (E < grid.energy.front()) {
        i_grid = 0;
      } else if (E > grid.energy.back()) {
        i_grid = grid.energy.size() - 2;
      } else {
        // Determine bounding indices based on which equal log-spaced
        // interval the energy is in
        int i_low = __ldg(&grid.grid_index[i_log_union]);
        int i_high = __ldg(&grid.grid_index[i_log_union + 1]) + 1;

        // Perform binary search over reduced range
        i_grid = i_low + lower_bound_index_linear(
                           &grid.energy[i_low], &grid.energy[i_high], E);
      }
      const auto& xs_left {nuclide.xs_[i_temp][i_grid]};
      const auto& xs_right {nuclide.xs_[i_temp][i_grid + 1]};
      // check for rare case where two energy points are the same
      if (grid.energy[i_grid] == grid.energy[i_grid + 1])
        ++i_grid;

      // calculate interpolation factor
      f = (E - grid.energy[i_grid]) /
          (grid.energy[i_grid + 1] - grid.energy[i_grid]);

      micro->index_temp = i_temp;
      micro->index_grid = i_grid;
      micro->interp_factor = f;

      // Calculate all microscopic cross sections
      micro->total = (1.0 - f) * xs_left.total + f * xs_right.total;
      micro->absorption =
        (1.0 - f) * xs_left.absorption + f * xs_right.absorption;

      if (nuclide.fissionable_) {
        // Calculate microscopic nuclide total cross section
        micro->fission = (1.0 - f) * xs_left.fission + f * xs_right.fission;

        // Calculate microscopic nuclide nu-fission cross section
        micro->nu_fission =
          (1.0 - f) * xs_left.nu_fission + f * xs_right.nu_fission;
      } else {
        micro->fission = 0.0;
        micro->nu_fission = 0.0;
      }

      // Calculate microscopic nuclide photon production cross section
      micro->photon_prod =
        (1.0 - f) * xs_left.photon_production + f * xs_right.photon_production;

      micro->index_sab = C_NONE;
      micro->sab_frac = 0.0;
      micro->last_E = E;
      micro->last_sqrtkT = p->sqrtkT_;
    }

    double const& atom_density = m.atom_density_[i];
    p->macro_xs_.total += atom_density * micro->total;
    p->macro_xs_.absorption += atom_density * micro->absorption;
    p->macro_xs_.fission += atom_density * micro->fission;
    p->macro_xs_.nu_fission += atom_density * micro->nu_fission;
  }
}

} // namespace gpu
} // namespace openmc
